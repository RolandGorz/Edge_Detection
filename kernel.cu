#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "opencv2/opencv.hpp"
#include <Windows.h>
#include <time.h>

using namespace cv;
using namespace std;

//texture that stores the input image data

texture<uchar, 2, hipReadModeElementType> src;

//bools that keep track if the user wants to save the outputs or an error occurred.

bool saveimage;
bool savevideo;
bool record;
bool failedOutput;
bool nocam;
bool fpsfail;

/*5x5 disk structuring element = {0, 1, 1, 1, 0},
								 {1, 1, 1, 1, 1},
								 {1, 1, 1, 1, 1},
								 {1, 1, 1, 1, 1},
								 {0, 1, 1, 1, 0}*/

__global__ void laplacian_texture(uchar *dev_lap, int rows, int cols) {
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	int tidy = threadIdx.y + blockIdx.y*blockDim.y;

	uchar max = 0;
	uchar min = 255;

	if (tidx >= cols || tidy >= rows) {
		return;
	}

	//loop through the 25 elements that the structuring element covers and keep track of the maximum and minimum value;

	for (int i = tidy - 2; i <= tidy + 2; i++) {
		for (int j = tidx - 6; j <= tidx + 6; j += 3) {
			if (i < 0 || i >= rows || j < 0 || j >= cols || ((i == tidy - 2) && (j == tidx - 6)) || ((i == tidy - 2) && (j == tidx + 6)) || ((i == tidy + 2) && (j == tidx - 6)) || ((i == tidy + 2) && (j == tidx + 6))) {
				continue;
			}

			uchar current = tex2D(src, j, i);

			if (current > max) {
				max = current;
			}
			if (current < min) {
				min = current;
			}
		}
	}

	//perform the laplacian at the current pixel

	uchar original = tex2D(src, tidx, tidy);

	if ((max - original) < (original - min)) {
		dev_lap[tidy * cols + tidx] = 0;
	}
	else {
		dev_lap[tidy * cols + tidx] = (max - original) - (original - min);
	}
}

__global__ void laplacian_simple(uchar *dev_data, uchar *dev_lap, int total_pixels, int cols) {

	//threadID provides every thread that runs on the GPU an individual value. Every thread works on a pixel in each color channel.

	int threadID = threadIdx.x + blockIdx.x * blockDim.x;
	uchar max = 0;
	uchar min = 255;

	//Since the arrays are 1 dimensional the right_edge and left_edge make sure a pixel from a seperate row is not accessed.

	int right_edge = 0;
	int left_edge = 0;

	/*If the image has more pixels than total threads running on the GPU then the thread also works on the next pixel that
	would have been missed*/

	for (threadID; threadID < total_pixels; threadID += blockDim.x * gridDim.x) {
		for (int row = threadID - (2 * cols); row <= threadID + (2 * cols); row += cols) {
			right_edge = cols * ((row / cols) + 1);
			left_edge = cols * (row / cols);
			for (int pos = row - 6; pos <= row + 6; pos+=3) {
				if (row < 0 || row >= total_pixels || pos < left_edge || pos >= right_edge || ((row == threadID - (2 * cols)) && (pos == row - 6)) || ((row == threadID - (2 * cols)) && (pos == row + 6)) || ((row == threadID + (2 * cols)) && (pos == row - 6)) || ((row == threadID + (2 * cols)) && (pos == row + 6))) {
					continue;
				}
				//Calculates the maximum and minimum within the area that the structuring element covers at the current pixel.

				uchar current = dev_data[pos];

				if (current > max) {
					max = current;
				}
				if (current < min) {
					min = current;
				}
			}
		}
		/*Calculates the dilation - the erosion of the current pixel to get the laplacian.
		If the dilation is less than the erosion then the pixel is set to 0 to prevent an overflow*/

		uchar original = dev_data[threadID];

		if ((max - original) < (original - min)) {
			dev_lap[threadID] = 0;
		}
		else {
			dev_lap[threadID] = (max - original) - (original - min);
		}

		//Reset the maximum and minimum storage for the next pixel

		max = 0;
		min = 255;
	}
}

//Used when the user inputs a video file but does not want to save the output
void videoNoSave() {

	//code to make the open file dialog box appear

	OPENFILENAME ofn;       // common dialog box structure
	char szFile[520];       // buffer for file name
	HWND hwnd = NULL;       // owner window
	HANDLE hf;              // file handle

							// Initialize OPENFILENAME
	ZeroMemory(&ofn, sizeof(ofn));
	ofn.lStructSize = sizeof(ofn);
	ofn.hwndOwner = hwnd;
	ofn.lpstrFile = szFile;
	// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
	// use the contents of szFile to initialize itself.
	ofn.lpstrFile[0] = '\0';
	ofn.nMaxFile = sizeof(szFile);
	ofn.lpstrFilter = "*.avi, *.divx\0*.avi;*.divx;\0\0*\0\0";
	ofn.nFilterIndex = 1;
	ofn.lpstrFileTitle = NULL;
	ofn.nMaxFileTitle = 0;
	ofn.lpstrInitialDir = ".";
	ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

	// Display the Open dialog box. 

	if (GetOpenFileName(&ofn) == TRUE)
		hf = CreateFile(ofn.lpstrFile,
			GENERIC_READ,
			0,
			(LPSECURITY_ATTRIBUTES)NULL,
			OPEN_EXISTING,
			FILE_ATTRIBUTE_NORMAL,
			(HANDLE)NULL);

	if (strlen(ofn.lpstrFile) == 0) {
		return;
	}

	for (int i = 0, int j = 0; i <= strlen(ofn.lpstrFile); i++, j++) {
		if (ofn.lpstrFile[i] == '\\') {
			ofn.lpstrFile[i] = '/';
		}
	}

	//close the handle because the open file dialog box had a handle on the file which would not allow videocapture to read it

	CloseHandle(hf);

	VideoCapture cap(ofn.lpstrFile);

	double fps = cap.get(CV_CAP_PROP_FPS);

	Mat frame;
	Mat lap_frame;

	namedWindow("Laplacian", 1);
	namedWindow("Original", 1);
	HWND LAPhwnd = (HWND)cvGetWindowHandle("Laplacian");
	HWND ORIhwnd = (HWND)cvGetWindowHandle("Original");

	hipArray *dev_data;

	uchar *dev_lap;

	dim3 gridsize, blocksize;

	/*Clamp address mode means that if a value that is outside of the texture array is accessed then instead of 
	seg faulting the nearest value along the endge is looked at. This is great for this program because the elements
	along that would already be part of the structuring element*/

	src.addressMode[0] = hipAddressModeClamp;
	src.addressMode[1] = hipAddressModeClamp;

	if (cap.isOpened() && IsWindowVisible(LAPhwnd)) {
		//malloc and calculate constants here to refrain from taking up time during the video loop.
		cap >> frame;
		lap_frame = frame.clone();

		blocksize.x = 32;
		blocksize.y = 32;
		gridsize.x = ceil(float(3 * frame.cols) / blocksize.x);
		gridsize.y = ceil(float(frame.rows) / blocksize.y);

		hipMallocArray(&dev_data, &src.channelDesc, 3 * frame.cols, frame.rows);

		hipMalloc((void**)&dev_lap, 3 * frame.rows * frame.cols * sizeof(uchar));
	}

	int size = 3 * frame.cols * frame.rows * sizeof(uchar);

	while (cap.isOpened() && IsWindowVisible(LAPhwnd)) {

		//Allow the user to close the original video, but keep playing the morphological operation.
		//If the user closes the laplacian video then close the rest of the windows as well.

		if (IsWindowVisible(ORIhwnd)) {
			imshow("Original", frame);
		}

		hipMemcpyToArray(dev_data, 0, 0, frame.data, size, hipMemcpyHostToDevice);

		hipBindTextureToArray(src, dev_data, src.channelDesc);

		laplacian_texture << <gridsize, blocksize >> >(dev_lap, frame.rows, 3 * frame.cols);

		hipMemcpy(lap_frame.data, dev_lap, size, hipMemcpyDeviceToHost);

		imshow("Laplacian", lap_frame);
		waitKey(1000 / fps);
		cap >> frame; // get a new frame from camera
		
		//If we reached the end of the video then clean up.

		if (frame.empty()) {
			destroyAllWindows();
			break;
		}
	}

	//If the laplacian window was closed then close the original as well

	if (IsWindowVisible(ORIhwnd)) {
		destroyAllWindows();
	}

	hipUnbindTexture(src);

	hipFree(dev_data);
	hipFree(dev_lap);
	cap.release();
}

//Very similar to video without save except for the fact that this one has saving involved

void videoSave() {
	OPENFILENAME ofn;       // common dialog box structure
	char szFile[520];       // buffer for file name
	HWND hwnd = NULL;       // owner window
	HANDLE hf;              // file handle

							// Initialize OPENFILENAME
	ZeroMemory(&ofn, sizeof(ofn));
	ofn.lStructSize = sizeof(ofn);
	ofn.hwndOwner = hwnd;
	ofn.lpstrFile = szFile;
	// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
	// use the contents of szFile to initialize itself.
	ofn.lpstrFile[0] = '\0';
	ofn.nMaxFile = sizeof(szFile);
	ofn.lpstrFilter = "*.avi, *.divx\0*.avi;*.divx;\0\0*\0\0";
	ofn.nFilterIndex = 1;
	ofn.lpstrFileTitle = NULL;
	ofn.nMaxFileTitle = 0;
	ofn.lpstrInitialDir = ".";
	ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

	// Display the Open dialog box. 

	if (GetOpenFileName(&ofn) == TRUE)
		hf = CreateFile(ofn.lpstrFile,
			GENERIC_READ,
			0,
			(LPSECURITY_ATTRIBUTES)NULL,
			OPEN_EXISTING,
			FILE_ATTRIBUTE_NORMAL,
			(HANDLE)NULL);

	if (strlen(ofn.lpstrFile) == 0) {
		return;
	}

	for (int i = 0, int j = 0; i <= strlen(ofn.lpstrFile); i++, j++) {
		if (ofn.lpstrFile[i] == '\\') {
			ofn.lpstrFile[i] = '/';
		}
	}

	CloseHandle(hf);

	VideoCapture cap(ofn.lpstrFile);
	Mat frame;
	Mat lap_frame;

	OPENFILENAME sfn;
	char syFile[520];
	ZeroMemory(&sfn, sizeof(sfn));
	sfn.lStructSize = sizeof(sfn);
	sfn.hwndOwner = NULL;
	sfn.lpstrFile = syFile;
	sfn.lpstrFile[0] = '\0';
	sfn.nMaxFile = sizeof(syFile);
	sfn.lpstrFilter = "*.avi\0*.avi;\0\0*\0";
	sfn.nFilterIndex = 1;
	sfn.lpstrFileTitle = NULL;
	sfn.nMaxFileTitle = 0;
	sfn.lpstrInitialDir = ".";
	sfn.Flags = OFN_PATHMUSTEXIST | OFN_OVERWRITEPROMPT | OFN_EXPLORER | OFN_ENABLEHOOK;
	sfn.lpstrDefExt = "avi";

	if (GetSaveFileName(&sfn) != true)
	{
		//do nothing
	}
	else {
		for (int i = 0, int j = 0; i <= strlen(sfn.lpstrFile); i++, j++) {
			if (sfn.lpstrFile[i] == '\\') {
				sfn.lpstrFile[i] = '/';
			}
		}
		remove(sfn.lpstrFile);

		double fps = cap.get(CV_CAP_PROP_FPS);

		VideoWriter output_cap(sfn.lpstrFile, -1, fps, Size(cap.get(CV_CAP_PROP_FRAME_WIDTH), cap.get(CV_CAP_PROP_FRAME_HEIGHT)));

		if (!output_cap.isOpened())
		{
			failedOutput = true;
			return;
		}

		namedWindow("Laplacian", 1);
		namedWindow("Original", 1);
		HWND LAPhwnd = (HWND)cvGetWindowHandle("Laplacian");
		HWND ORIhwnd = (HWND)cvGetWindowHandle("Original");

		hipArray *dev_data;

		uchar *dev_lap;

		dim3 gridsize, blocksize;

		src.addressMode[0] = hipAddressModeClamp;
		src.addressMode[1] = hipAddressModeClamp;

		if (cap.isOpened() && IsWindowVisible(LAPhwnd)) {
			cap >> frame;
			lap_frame = frame.clone();

			blocksize.x = 32;
			blocksize.y = 32;
			gridsize.x = ceil(float(3 * frame.cols) / blocksize.x);
			gridsize.y = ceil(float(frame.rows) / blocksize.y);

			hipMallocArray(&dev_data, &src.channelDesc, 3 * frame.cols, frame.rows);

			hipMalloc((void**)&dev_lap, 3 * frame.rows * frame.cols * sizeof(uchar));
		}

		int size = 3 * frame.cols * frame.rows * sizeof(uchar);

		while (cap.isOpened() && IsWindowVisible(LAPhwnd)) {

			if (IsWindowVisible(ORIhwnd)) {
				imshow("Original", frame);
			}

			hipMemcpyToArray(dev_data, 0, 0, frame.data, size, hipMemcpyHostToDevice);

			hipBindTextureToArray(src, dev_data, src.channelDesc);

			laplacian_texture << <gridsize, blocksize >> >(dev_lap, frame.rows, 3 * frame.cols);

			hipMemcpy(lap_frame.data, dev_lap, size, hipMemcpyDeviceToHost);

			imshow("Laplacian", lap_frame);
			output_cap.write(lap_frame);
			waitKey(1000 / fps);
			cap >> frame; // get a new frame from camera
			if (frame.empty()) {
				destroyAllWindows();
				break;
			}
		}

		if (IsWindowVisible(ORIhwnd)) {
			destroyAllWindows();
		}

		hipUnbindTexture(src);

		hipFree(dev_data);
		hipFree(dev_lap);
	}
	cap.release();
}

void camera_feed_nosave() {
	VideoCapture cap(0); // open the default camera
	if (!cap.isOpened()) {
		nocam = true;
		cout << "Failed to find default camera" << endl;
		return;
	}
	//Let the user set camera resolution
	//cap.set(CV_CAP_PROP_FRAME_WIDTH, 640);
	//cap.set(CV_CAP_PROP_FRAME_HEIGHT, 480);

	//Find the camera fps here

	int num_frames = 120;
	time_t start, end;
	Mat frame;
	Mat lap_frame;

	cap >> frame;
	if (frame.empty()) {
		nocam = true;
		cout << "Failed to find default camera" << endl;
		return;
	}

	time(&start);
	for (int i = 0; i < num_frames; i++) {
		cap >> frame;
	}
	time(&end);
	double seconds = difftime(end, start);

	if (seconds == 0) {
		cout << "Error with camera. Failed to calculate fps" << endl;
		return;
	}

	double fps = num_frames / seconds;

	cout << fps << endl;

	namedWindow("Laplacian", 1);
	namedWindow("Original", 1);
	HWND LAPhwnd = (HWND)cvGetWindowHandle("Laplacian");
	HWND ORIhwnd = (HWND)cvGetWindowHandle("Original");

	hipArray *dev_data;

	uchar *dev_lap;

	dim3 gridsize, blocksize;

	src.addressMode[0] = hipAddressModeClamp;
	src.addressMode[1] = hipAddressModeClamp;

	if (IsWindowVisible(LAPhwnd)) {
		cap >> frame;
		lap_frame = frame.clone();

		blocksize.x = 32;
		blocksize.y = 32;
		gridsize.x = ceil(float(3 * frame.cols) / blocksize.x);
		gridsize.y = ceil(float(frame.rows) / blocksize.y);

		hipMallocArray(&dev_data, &src.channelDesc, 3 * frame.cols, frame.rows);

		hipMalloc((void**)&dev_lap, 3 * frame.rows * frame.cols * sizeof(uchar));
	}

	int size = 3 * frame.cols * frame.rows * sizeof(uchar);

	while (IsWindowVisible(LAPhwnd)) {
		if (IsWindowVisible(ORIhwnd)) {
			imshow("Original", frame);
		}

		hipMemcpyToArray(dev_data, 0, 0, frame.data, size, hipMemcpyHostToDevice);

		hipBindTextureToArray(src, dev_data, src.channelDesc);

		laplacian_texture << <gridsize, blocksize >> >(dev_lap, frame.rows, 3 * frame.cols);

		hipMemcpy(lap_frame.data, dev_lap, size, hipMemcpyDeviceToHost);

		imshow("Laplacian", lap_frame);
		waitKey(1000 / fps);
		cap >> frame; // get a new frame from camera
	}

	if (IsWindowVisible(ORIhwnd)) {
		destroyAllWindows();
	}

	hipUnbindTexture(src);

	hipFree(dev_data);
	hipFree(dev_lap);
	cap.release();
}

void camera_feed_save() {
	VideoCapture cap(0);// open the default camera
	if (!cap.isOpened()) {
		nocam = true;
		cout << "Failed to find default camera" << endl;
		return;
	}
	//Let the user set camera resolution
	//cap.set(CV_CAP_PROP_FRAME_WIDTH, 640);
	//cap.set(CV_CAP_PROP_FRAME_HEIGHT, 480);
	
	OPENFILENAME sfn;
	char syFile[520];
	ZeroMemory(&sfn, sizeof(sfn));
	sfn.lStructSize = sizeof(sfn);
	sfn.hwndOwner = NULL;
	sfn.lpstrFile = syFile;
	sfn.lpstrFile[0] = '\0';
	sfn.nMaxFile = sizeof(syFile);
	sfn.lpstrFilter = "*.avi\0*.avi;\0\0*\0";
	sfn.nFilterIndex = 1;
	sfn.lpstrFileTitle = NULL;
	sfn.nMaxFileTitle = 0;
	sfn.lpstrInitialDir = ".";
	sfn.Flags = OFN_PATHMUSTEXIST | OFN_OVERWRITEPROMPT | OFN_EXPLORER | OFN_ENABLEHOOK;
	sfn.lpstrDefExt = "avi";

	//Find the camera fps here

	int num_frames = 120;
	time_t start, end;
	Mat frame;
	Mat lap_frame;

	cap >> frame;
	if (frame.empty()) {
		nocam = true;
		cout << "Failed to find default camera" << endl;
		return;
	}

	time(&start);
	for (int i = 0; i < num_frames; i++) {
		cap >> frame;
	}
	time(&end);
	double seconds = difftime(end, start);

	if (seconds == 0) {
		fpsfail = true;
		cout << "Error with camera. Failed to calculate fps" << endl;
		return;
	}

	double fps = num_frames / seconds;

	cout << fps << endl;

	if (GetSaveFileName(&sfn) != true)
	{
		//do nothing
	}
	else {
		for (int i = 0, int j = 0; i <= strlen(sfn.lpstrFile); i++, j++) {
			if (sfn.lpstrFile[i] == '\\') {
				sfn.lpstrFile[i] = '/';
			}
		}
		remove(sfn.lpstrFile);

		//cap.get(CV_CAP_PROP_FPS) is used for input videos not webcam.
		VideoWriter output_cap(sfn.lpstrFile, -1, fps, Size(cap.get(CV_CAP_PROP_FRAME_WIDTH), cap.get(CV_CAP_PROP_FRAME_HEIGHT)));
		if (!output_cap.isOpened())
		{
			failedOutput = true;
			return;
		}

		namedWindow("Laplacian", 1);
		namedWindow("Original", 1);
		HWND LAPhwnd = (HWND)cvGetWindowHandle("Laplacian");
		HWND ORIhwnd = (HWND)cvGetWindowHandle("Original");

		hipArray *dev_data;

		uchar *dev_lap;

		dim3 gridsize, blocksize;

		src.addressMode[0] = hipAddressModeClamp;
		src.addressMode[1] = hipAddressModeClamp;

		if (IsWindowVisible(LAPhwnd)) {
			cap >> frame;
			lap_frame = frame.clone();

			blocksize.x = 32;
			blocksize.y = 32;
			gridsize.x = ceil(float(3 * frame.cols) / blocksize.x);
			gridsize.y = ceil(float(frame.rows) / blocksize.y);

			hipMallocArray(&dev_data, &src.channelDesc, 3 * frame.cols, frame.rows);

			hipMalloc((void**)&dev_lap, 3 * frame.rows * frame.cols * sizeof(uchar));
		}

		int size = 3 * frame.cols * frame.rows * sizeof(uchar);

		while (IsWindowVisible(LAPhwnd)) {
			if (IsWindowVisible(ORIhwnd)) {
				imshow("Original", frame);
			}

			hipMemcpyToArray(dev_data, 0, 0, frame.data, size, hipMemcpyHostToDevice);

			hipBindTextureToArray(src, dev_data, src.channelDesc);

			laplacian_texture << <gridsize, blocksize >> >(dev_lap, frame.rows, 3 * frame.cols);

			hipMemcpy(lap_frame.data, dev_lap, size, hipMemcpyDeviceToHost);

			imshow("Laplacian", lap_frame);
			output_cap.write(lap_frame);
			waitKey(1000 / fps);
			cap >> frame; // get a new frame from camera
		}

		if (IsWindowVisible(ORIhwnd)) {
			destroyAllWindows();
		}

		hipUnbindTexture(src);

		hipFree(dev_data);
		hipFree(dev_lap);
	}
	cap.release();
}

void image_texture() {

	//Read the filename that the user wishes to enter and keep asking for user input until a file can be opened or the user quits

	OPENFILENAME ofn;       // common dialog box structure
	char szFile[520];       // buffer for file name
	HWND hwnd = NULL;       // owner window
	HANDLE hf;              // file handle

							// Initialize OPENFILENAME
	ZeroMemory(&ofn, sizeof(ofn));
	ofn.lStructSize = sizeof(ofn);
	ofn.hwndOwner = hwnd;
	ofn.lpstrFile = szFile;
	// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
	// use the contents of szFile to initialize itself.
	ofn.lpstrFile[0] = '\0';
	ofn.nMaxFile = sizeof(szFile);
	ofn.lpstrFilter = "*.jpg, *.png, *.bmp, *.dib, *.jpeg, *.jpe, *.jfif, *.tif, *.tiff\0*.jpg;*.png;*.bmp;*.dib;*.jpeg;*.jpe;*.jfif;*.tif;*.tiff\0\0*\0\0\0\0\0\0\0\0\0";
	ofn.nFilterIndex = 1;
	ofn.lpstrFileTitle = NULL;
	ofn.nMaxFileTitle = 0;
	ofn.lpstrInitialDir = ".";
	ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

	// Display the Open dialog box. 

	if (GetOpenFileName(&ofn) == TRUE)
		hf = CreateFile(ofn.lpstrFile,
			GENERIC_READ,
			0,
			(LPSECURITY_ATTRIBUTES)NULL,
			OPEN_EXISTING,
			FILE_ATTRIBUTE_NORMAL,
			(HANDLE)NULL);

	if (strlen(ofn.lpstrFile) == 0) {
		return;
	}

	for (int i = 0, int j = 0; i <= strlen(ofn.lpstrFile); i++, j++) {
		if (ofn.lpstrFile[i] == '\\') {
			ofn.lpstrFile[i] = '/';
		}
	}

	CloseHandle(hf);

	Mat image = imread(ofn.lpstrFile, 1);

	namedWindow("INPUT", CV_WINDOW_KEEPRATIO);
	imshow("INPUT", image);

	uchar *dev_lap;

	hipMalloc((void**)&dev_lap, 3 * image.rows * image.cols * sizeof(uchar));

	hipArray *dev_data;

	hipMallocArray(&dev_data, &src.channelDesc, 3 * image.cols, image.rows);

	hipMemcpyToArray(dev_data, 0, 0, image.data, 3 * image.cols * image.rows * sizeof(uchar), hipMemcpyHostToDevice);

	hipBindTextureToArray(src, dev_data, src.channelDesc);

	dim3 gridsize, blocksize;
	blocksize.x = 32;
	blocksize.y = 32;
	gridsize.x = ceil(float(3 * image.cols) / blocksize.x);
	gridsize.y = ceil(float(image.rows) / blocksize.y);

	laplacian_texture <<<gridsize, blocksize>>>(dev_lap, image.rows, 3 * image.cols);

	hipMemcpy(image.data, dev_lap, 3 * image.rows * image.cols * sizeof(uchar), hipMemcpyDeviceToHost);

	hipUnbindTexture(src);

	hipFree(dev_data);
	hipFree(dev_lap);

	namedWindow("OUTPUT", CV_WINDOW_KEEPRATIO);
	imshow("OUTPUT", image);

	if (saveimage) {
		OPENFILENAME sfn;
		char syFile[520];
		ZeroMemory(&sfn, sizeof(sfn));
		sfn.lStructSize = sizeof(sfn);
		sfn.hwndOwner = NULL;
		sfn.lpstrFile = syFile;
		sfn.lpstrFile[0] = '\0';
		sfn.nMaxFile = sizeof(syFile);
		sfn.lpstrFilter = "*.jpg, *.png, *.bmp, *.dib, *.jpeg, *.jpe, *.jfif, *.tif, *.tiff\0*.jpg;*.png;*.bmp;*.dib;*.jpeg;*.jpe;*.jfif;*.tif;*.tiff\0\0*\0\0\0\0\0\0\0\0\0";
		sfn.nFilterIndex = 1;
		sfn.lpstrFileTitle = NULL;
		sfn.nMaxFileTitle = 0;
		sfn.lpstrInitialDir = ".";
		sfn.Flags = OFN_PATHMUSTEXIST | OFN_OVERWRITEPROMPT | OFN_EXPLORER | OFN_ENABLEHOOK;
		sfn.lpstrDefExt = "jpg";

		if (GetSaveFileName(&sfn) != true)
		{
			//do nothing
		}
		else {
			for (int i = 0, int j = 0; i <= strlen(sfn.lpstrFile); i++, j++) {
				if (sfn.lpstrFile[i] == '\\') {
					sfn.lpstrFile[i] = '/';
				}
			}
			imwrite(sfn.lpstrFile, image);
		}
	}
	waitKey(0);
	return;
}

void image_simple() {

	//Read the filename that the user wishes to enter and keep asking for user input until a file can be opened or the user quits

	OPENFILENAME ofn;       // common dialog box structure
	char szFile[520];       // buffer for file name
	HWND hwnd = NULL;       // owner window
	HANDLE hf;              // file handle

							// Initialize OPENFILENAME
	ZeroMemory(&ofn, sizeof(ofn));
	ofn.lStructSize = sizeof(ofn);
	ofn.hwndOwner = hwnd;
	ofn.lpstrFile = szFile;
	// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
	// use the contents of szFile to initialize itself.
	ofn.lpstrFile[0] = '\0';
	ofn.nMaxFile = sizeof(szFile);
	ofn.lpstrFilter = "Supported Image Files\0*.jpg;*.png;*.bmp;*.dib;*.jpeg;*.jpe;*.jfif;*.tif;*.tiff\0ALL FILES\0*\0\0\0\0\0\0\0\0\0";
	ofn.nFilterIndex = 1;
	ofn.lpstrFileTitle = NULL;
	ofn.nMaxFileTitle = 0;
	ofn.lpstrInitialDir = NULL;
	ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

	// Display the Open dialog box. 

	if (GetOpenFileName(&ofn) == TRUE)
		hf = CreateFile(ofn.lpstrFile,
			GENERIC_READ,
			0,
			(LPSECURITY_ATTRIBUTES)NULL,
			OPEN_EXISTING,
			FILE_ATTRIBUTE_NORMAL,
			(HANDLE)NULL);

	if (strlen(ofn.lpstrFile) == 0) {
		return;
	}

	for (int i = 0, int j = 0; i <= strlen(ofn.lpstrFile); i++, j++) {
		if (ofn.lpstrFile[i] == '\\') {
			ofn.lpstrFile[i] = '/';
		}
	}

	CloseHandle(hf);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	Mat image = imread(ofn.lpstrFile, 1);

	namedWindow("INPUT", CV_WINDOW_KEEPRATIO);
	imshow("INPUT", image);

	/*Split the image into the 3 image channels Blue, Green, and Red respectively. This makes 3 arrays that contain the intensity values
	of each image channel. These arrays are then allocated and passed to the GPU. LapB contains the intensity values after the algorithm
	for computing the laplacian completes.*/

	uchar *dev_data;
	uchar *dev_lap;

	hipMalloc((void**)&dev_data, image.rows * image.cols * 3 * sizeof(uchar));
	hipMalloc((void**)&dev_lap, image.rows * image.cols * 3 * sizeof(uchar));

	hipMemcpy(dev_data, image.data, image.rows * image.cols * 3 * sizeof(uchar), hipMemcpyHostToDevice);

	/*Call the CUDA kernel with a grid size of 512 that each one will be run on a Streaming Multiprocessor with each
	Multiprocessor running 1024 threads*/

	laplacian_simple << <512, 1024 >> >(dev_data, dev_lap, 3 * image.rows * image.cols, 3 * image.cols);
	//Transfer the lapB array from the device to the host

	hipMemcpy(image.data, dev_lap, image.rows * image.cols * 3 * sizeof(uchar), hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << endl;

	hipFree(dev_data);
	hipFree(dev_lap);

	//Merge the 3 seperate channel arrays into one array for the output image. Display the input and output image.

	namedWindow("OUTPUT", CV_WINDOW_KEEPRATIO);
	imshow("OUTPUT", image);

	OPENFILENAME sfn;
	char syFile[520];
	ZeroMemory(&sfn, sizeof(sfn));
	sfn.lStructSize = sizeof(sfn);
	sfn.hwndOwner = NULL;
	sfn.lpstrFile = syFile;
	sfn.lpstrFile[0] = '\0';
	sfn.nMaxFile = sizeof(syFile);
	sfn.lpstrFilter = "*.jpg, *.png, *.bmp, *.dib, *.jpeg, *.jpe, *.jfif, *.tif, *.tiff\0*.jpg;*.png;*.bmp;*.dib;*.jpeg;*.jpe;*.jfif;*.tif;*.tiff\0\0*\0\0\0\0\0\0\0\0\0";
	sfn.nFilterIndex = 1;
	sfn.lpstrFileTitle = NULL;
	sfn.nMaxFileTitle = 0;
	sfn.lpstrInitialDir = ".";
	sfn.Flags = OFN_PATHMUSTEXIST | OFN_OVERWRITEPROMPT | OFN_EXPLORER | OFN_ENABLEHOOK;
	sfn.lpstrDefExt = "jpg";

	if (GetSaveFileName(&sfn) != true)
	{
		cout << "Saving file canceled, closing program in 10 secconds." << endl;
	}
	else {
		for (int i = 0, int j = 0; i <= strlen(sfn.lpstrFile); i++, j++) {
			if (sfn.lpstrFile[i] == '\\') {
				sfn.lpstrFile[i] = '/';
			}
		}
		imwrite(sfn.lpstrFile, image);
	}
	waitKey(0);
	return;
}

/* This is where all the input to the window goes to */
HWND button1;
HWND check1;
HWND button2;
HWND check2;
HWND button3;
HWND check3;
char input[520];
HWND edit;
HWND text;
LRESULT CALLBACK WndProc(HWND hwnd, UINT Message, WPARAM wParam, LPARAM lParam) {
	switch (Message) {
		
		case WM_CREATE: {

			text = CreateWindow(TEXT("STATIC"), TEXT("Laplacian Morphological Operation"), 
				WS_VISIBLE | WS_CHILD, 
				190, 10, 
				400, 25, 
				hwnd, (HMENU) NULL, NULL, NULL);

			HFONT text_change = CreateFont(20, 0, 0, 0, FW_DONTCARE, FALSE, FALSE, FALSE, ANSI_CHARSET, OUT_DEFAULT_PRECIS, CLIP_DEFAULT_PRECIS, DEFAULT_QUALITY, DEFAULT_PITCH | FF_SWISS, "Times New Roman");
			SendMessage(text, WM_SETFONT, WPARAM(text_change), TRUE);

			//GetWindowText(edit, input, 260);

			text = CreateWindow(TEXT("STATIC"), TEXT("Live input needs a few seconds to calculate the camera's FPS. Please wait after selecting."),
				WS_VISIBLE | WS_CHILD,
				190, 255,
				400, 50,
				hwnd, (HMENU)NULL, NULL, NULL);

			button1 = CreateWindow(TEXT("BUTTON"), TEXT("Image Input"),
				WS_VISIBLE | WS_CHILD,
				10, 50,
				150, 50,
				hwnd, (HMENU) 1, NULL, NULL);

			text_change = CreateFont(30, 10, 0, 0, FW_DONTCARE, FALSE, FALSE, FALSE, ANSI_CHARSET, OUT_DEFAULT_PRECIS, CLIP_DEFAULT_PRECIS, DEFAULT_QUALITY, DEFAULT_PITCH | FF_SWISS, "Arial");
			SendMessage(button1, WM_SETFONT, WPARAM(text_change), TRUE);

			check1 = CreateWindow(TEXT("button"), TEXT("Save Image"),
				WS_VISIBLE | WS_CHILD | BS_CHECKBOX,
				20, 100, 
				100, 20,
				hwnd, (HMENU)2, ((LPCREATESTRUCT)lParam)->hInstance, NULL);
			CheckDlgButton(hwnd, 2, BST_CHECKED);

			button2 = CreateWindow(TEXT("BUTTON"), TEXT("Video Input"),
				WS_VISIBLE | WS_CHILD,
				10, 150,
				150, 50,
				hwnd, (HMENU)3, NULL, NULL);

			check2 = CreateWindow(TEXT("button"), TEXT("Save Video"),
				WS_VISIBLE | WS_CHILD | BS_CHECKBOX,
				20, 200,
				95, 20,
				hwnd, (HMENU)4, ((LPCREATESTRUCT)lParam)->hInstance, NULL);
			CheckDlgButton(hwnd, 4, BST_UNCHECKED);

			text_change = CreateFont(30, 10, 0, 0, FW_DONTCARE, FALSE, FALSE, FALSE, ANSI_CHARSET, OUT_DEFAULT_PRECIS, CLIP_DEFAULT_PRECIS, DEFAULT_QUALITY, DEFAULT_PITCH | FF_SWISS, "Arial");
			SendMessage(button2, WM_SETFONT, WPARAM(text_change), TRUE);

			button3 = CreateWindow(TEXT("BUTTON"), TEXT("Live Input"),
				WS_VISIBLE | WS_CHILD,
				10, 250,
				150, 50,
				hwnd, (HMENU)5, NULL, NULL);

			check3 = CreateWindow(TEXT("button"), TEXT("Record Video"),
				WS_VISIBLE | WS_CHILD | BS_CHECKBOX,
				20, 300,
				105, 20,
				hwnd, (HMENU)6, ((LPCREATESTRUCT)lParam)->hInstance, NULL);
			CheckDlgButton(hwnd, 6, BST_UNCHECKED);

			text_change = CreateFont(30, 10, 0, 0, FW_DONTCARE, FALSE, FALSE, FALSE, ANSI_CHARSET, OUT_DEFAULT_PRECIS, CLIP_DEFAULT_PRECIS, DEFAULT_QUALITY, DEFAULT_PITCH | FF_SWISS, "Arial");
			SendMessage(button3, WM_SETFONT, WPARAM(text_change), TRUE);

			break;
		}
		case WM_COMMAND: {

			if (LOWORD(wParam) == 1) {
				/*GetWindowText(edit, input, 260);
				MessageBox(hwnd, input, "title for popup", MB_ICONINFORMATION);*/
				if (IsDlgButtonChecked(hwnd, 2)) {
					saveimage = true;
				}
				else {
					saveimage = false;
				}
				EnableWindow(button1, false);
				EnableWindow(check1, false);
				EnableWindow(button2, false);
				EnableWindow(check2, false);
				EnableWindow(button3, false);
				EnableWindow(check3, false);
				image_texture();
				EnableWindow(button1, true);
				EnableWindow(check1, true);
				EnableWindow(button2, true);
				EnableWindow(check2, true);
				EnableWindow(button3, true);
				EnableWindow(check3, true);
			}

			if (LOWORD(wParam) == 2) {
				BOOL checked = IsDlgButtonChecked(hwnd, 2);
				if (checked) {
					CheckDlgButton(hwnd, 2, BST_UNCHECKED);
				}
				else {
					CheckDlgButton(hwnd, 2, BST_CHECKED);
				}
			}

			if (LOWORD(wParam) == 3) {
				/*GetWindowText(edit, input, 260);
				MessageBox(hwnd, input, "title for popup", MB_ICONINFORMATION);*/
				if (IsDlgButtonChecked(hwnd, 4)) {
					savevideo = true;
				}
				else {
					savevideo = false;
				}
				EnableWindow(button1, false);
				EnableWindow(check1, false);
				EnableWindow(button2, false);
				EnableWindow(check2, false);
				EnableWindow(button3, false);
				EnableWindow(check3, false);
				if (savevideo) {
					videoSave();
				}
				else {
					videoNoSave();
				}
				if (failedOutput) {
					MessageBox(hwnd, "Output video could not be opened use different compression option", "Error", MB_ICONINFORMATION);
				}
				failedOutput = false;
				EnableWindow(button1, true);
				EnableWindow(check1, true);
				EnableWindow(button2, true);
				EnableWindow(check2, true);
				EnableWindow(button3, true);
				EnableWindow(check3, true);
			}

			if (LOWORD(wParam) == 4) {
				BOOL checked = IsDlgButtonChecked(hwnd, 4);
				if (checked) {
					CheckDlgButton(hwnd, 4, BST_UNCHECKED);
				}
				else {
					CheckDlgButton(hwnd, 4, BST_CHECKED);
				}
			}

			if (LOWORD(wParam) == 5) {
				/*GetWindowText(edit, input, 260);
				MessageBox(hwnd, input, "title for popup", MB_ICONINFORMATION);*/
				if (IsDlgButtonChecked(hwnd, 6)) {
					record = true;
				}
				else {
					record = false;
				}
				EnableWindow(button1, false);
				EnableWindow(check1, false);
				EnableWindow(button2, false);
				EnableWindow(check2, false);
				EnableWindow(button3, false);
				EnableWindow(check3, false);
				if (record) {
					camera_feed_save();
				}
				else {
					camera_feed_nosave();
				}
				if (failedOutput) {
					MessageBox(hwnd, "Output video could not be opened use different compression option", "Error", MB_ICONINFORMATION);
				}
				if (nocam) {
					MessageBox(hwnd, "Failed to find default camera", "Error", MB_ICONINFORMATION);
				}
				if (fpsfail) {
					MessageBox(hwnd, "Error with camera. Failed to calculate fps", "Error", MB_ICONINFORMATION);
				}
				failedOutput = false;
				nocam = false;
				fpsfail = false;
				EnableWindow(button1, true);
				EnableWindow(check1, true);
				EnableWindow(button2, true);
				EnableWindow(check2, true);
				EnableWindow(button3, true);
				EnableWindow(check3, true);
			}

			if (LOWORD(wParam) == 6) {
				BOOL checked = IsDlgButtonChecked(hwnd, 6);
				if (checked) {
					CheckDlgButton(hwnd, 6, BST_UNCHECKED);
				}
				else {
					CheckDlgButton(hwnd, 6, BST_CHECKED);
				}
			}

			break;
		}

		/* Upon destruction, tell the main thread to stop */
		case WM_DESTROY: {
			PostQuitMessage(0);
			break;
		}

					 /* All other messages (a lot of them) are processed using default procedures */
		default:
			return DefWindowProc(hwnd, Message, wParam, lParam);
	}
	return 0;
}

/* The 'main' function of Win32 GUI programs: this is where execution starts */
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow) {
	WNDCLASSEX wc; /* A properties struct of our window */
	HWND hwnd; /* A 'HANDLE', hence the H, or a pointer to our window */
	MSG msg; /* A temporary location for all messages */

			 /* zero out the struct and set the stuff we want to modify */
	memset(&wc, 0, sizeof(wc));
	wc.cbSize = sizeof(WNDCLASSEX);
	wc.lpfnWndProc = WndProc; /* This is where we will send messages to */
	wc.hInstance = hInstance;
	wc.hCursor = LoadCursor(NULL, IDC_ARROW);

	/* White, COLOR_WINDOW is just a #define for a system color, try Ctrl+Clicking it */
	//wc.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
	wc.hbrBackground = GetSysColorBrush(COLOR_3DFACE);
	wc.lpszClassName = "WindowClass";

	wc.hIcon = (HICON)LoadImage( // returns a HANDLE so we have to cast to HICON
		NULL,             // hInstance must be NULL when loading from a file
		"lapIcon.ico",   // the icon file name
		IMAGE_ICON,       // specifies that the file is an icon
		0,                // width of the image (we'll specify default later on)
		0,                // height of the image
		LR_LOADFROMFILE |  // we want to load a file (as opposed to a resource)
		LR_DEFAULTSIZE |   // default metrics based on the type (IMAGE_ICON, 32x32)
		LR_SHARED         // let the system release the handle when it's no longer used
		);
	wc.hIconSm = LoadIcon(NULL, NULL); /* use the name "A" to use the project icon */

	if (!RegisterClassEx(&wc)) {
		MessageBox(NULL, "Window Registration Failed!", "Error!", MB_ICONEXCLAMATION | MB_OK);
		return 0;
	}

	hwnd = CreateWindowEx(WS_EX_CLIENTEDGE, "WindowClass", "CUDA Laplacian", WS_VISIBLE | WS_SYSMENU,
		CW_USEDEFAULT, /* x */
		CW_USEDEFAULT, /* y */
		640, /* width */
		480, /* height */
		NULL, NULL, hInstance, NULL);

	if (hwnd == NULL) {
		MessageBox(NULL, "Window Creation Failed!", "Error!", MB_ICONEXCLAMATION | MB_OK);
		return 0;
	}

	/*
	This is the heart of our program where all input is processed and
	sent to WndProc. Note that GetMessage blocks code flow until it receives something, so
	this loop will not produce unreasonably high CPU usage
	*/
	while (GetMessage(&msg, NULL, 0, 0) > 0) { /* If no error is received... */
		TranslateMessage(&msg); /* Translate key codes to chars if present */
		DispatchMessage(&msg); /* Send it to WndProc */
	}
	return msg.wParam;
}